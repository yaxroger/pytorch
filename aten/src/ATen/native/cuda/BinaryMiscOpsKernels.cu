#include "hip/hip_runtime.h"
#include <ATen/Dispatch.h>
#include <ATen/native/DispatchStub.h>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/native/TensorIterator.h>
#include <ATen/native/BinaryOps.h>


// NOTE: CUDA on Windows requires that the enclosing function
// of a __device__ lambda not have internal linkage.

namespace at { namespace native {

void atan2_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(iter.common_dtype(), "atan2_cuda", [&]() {
    gpu_kernel_with_scalars(iter, []GPU_LAMBDA(scalar_t a, scalar_t b) -> scalar_t {
      return ::atan2(a, b);
    });
  });
}

void bitwise_and_kernel_cuda(TensorIterator& iter) {
  if (iter.dtype() == ScalarType::Bool) {
    gpu_kernel_with_scalars(
        iter,
        []GPU_LAMBDA(bool a, bool b) {
          return a && b;
    });
  } else {
    AT_DISPATCH_INTEGRAL_TYPES(iter.dtype(), "bitwise_and_cuda", [&]() {
      gpu_kernel_with_scalars(
          iter,
          []GPU_LAMBDA(scalar_t a, scalar_t b) -> scalar_t {
            return a & b;
      });
    });
  }
}

void bitwise_or_kernel_cuda(TensorIterator& iter) {
  if (iter.dtype() == ScalarType::Bool) {
    gpu_kernel_with_scalars(
        iter,
        []GPU_LAMBDA(bool a, bool b) {
          return a || b;
    });
  } else {
    AT_DISPATCH_INTEGRAL_TYPES(iter.dtype(), "bitwise_or_cuda", [&]() {
      gpu_kernel_with_scalars(
          iter,
          []GPU_LAMBDA(scalar_t a, scalar_t b) -> scalar_t {
            return a | b;
      });
    });
  }
}

void bitwise_xor_kernel_cuda(TensorIterator& iter) {
  if (iter.dtype() == ScalarType::Bool) {
    // Boolean type does not work with ^ (bitwise XOR) in C++. bitwise_xor wraps this operation for both Boolean and
    // integral types.
    gpu_kernel_with_scalars(
          iter,
          []GPU_LAMBDA(bool a, bool b) {
            return a != b;
          });
  } else {
    AT_DISPATCH_INTEGRAL_TYPES(iter.dtype(), "bitwise_xor_cuda", [&]() {
      gpu_kernel_with_scalars(
          iter,
          []GPU_LAMBDA(scalar_t a, scalar_t b) -> scalar_t {
            return a ^ b;
      });
    });
  }
}

void lshift_kernel_cuda(TensorIterator& iter) {
  if (iter.dtype() == ScalarType::Float || iter.dtype() == ScalarType::Double) {
    AT_DISPATCH_FLOATING_TYPES(iter.dtype(), "lshift_cuda", [&]() {
      gpu_kernel_with_scalars(
        iter,
        []GPU_LAMBDA(scalar_t a, scalar_t b) -> scalar_t {
          return a * std::pow((scalar_t)(2), b);
      });
    });
  } else {
    AT_DISPATCH_INTEGRAL_TYPES(iter.dtype(), "lshift_cuda", [&]() {
      gpu_kernel_with_scalars(iter,
        []GPU_LAMBDA(scalar_t a, scalar_t b) -> scalar_t {
          return a << b;
      });
    });
  }
}

void logical_and_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_ALL_TYPES_AND2(kHalf, kBool, iter.common_dtype(), "logical_and_cuda", [&]() {
    gpu_kernel_with_scalars(iter, []GPU_LAMBDA(scalar_t a, scalar_t b) -> bool {
      return a && b;
    });
  });
}

void logical_or_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_ALL_TYPES_AND2(kHalf, kBool, iter.common_dtype(), "logical_or_cuda", [&]() {
    gpu_kernel_with_scalars(iter, []GPU_LAMBDA(scalar_t a, scalar_t b) -> bool {
      return a || b;
    });
  });
}

void logical_xor_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_ALL_TYPES_AND2(kHalf, kBool, iter.common_dtype(), "logical_xor_cuda", [&]() {
    gpu_kernel_with_scalars(iter, []GPU_LAMBDA(scalar_t a, scalar_t b) -> bool {
      return bool(a) != bool(b);
    });
  });
}

void smooth_l1_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_ALL_TYPES_AND(kHalf, iter.dtype(), "smooth_l1_cuda", [&]() {
    gpu_kernel(iter, [] GPU_LAMBDA(scalar_t a, scalar_t b) -> scalar_t {
      auto z = fabs(a - b);
      return z < scalar_t(1.) ? scalar_t(0.5) * z * z : z - scalar_t(0.5);
    });
  });
}

void sigmoid_backward_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(iter.dtype(), "sigmoid_backward_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a, scalar_t b) -> scalar_t {
      return a * (scalar_t(1.) - b) * b;
    });
  });
}

void tanh_backward_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(iter.dtype(), "tanh_backward_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a, scalar_t b) -> scalar_t {
      return a * (scalar_t(1.) - b * b);
    });
  });
}

void mse_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(iter.dtype(), "mse_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a, scalar_t b) -> scalar_t {
      auto diff = a - b;
      return diff * diff;
    });
  });
}

REGISTER_DISPATCH(atan2_stub, &atan2_kernel_cuda);
REGISTER_DISPATCH(bitwise_and_stub, &bitwise_and_kernel_cuda);
REGISTER_DISPATCH(bitwise_or_stub, &bitwise_or_kernel_cuda);
REGISTER_DISPATCH(lshift_stub, &lshift_kernel_cuda);
REGISTER_DISPATCH(bitwise_xor_stub, &bitwise_xor_kernel_cuda);
REGISTER_DISPATCH(logical_and_stub, &logical_and_kernel_cuda);
REGISTER_DISPATCH(logical_or_stub, &logical_or_kernel_cuda);
REGISTER_DISPATCH(logical_xor_stub, &logical_xor_kernel_cuda);
REGISTER_DISPATCH(smooth_l1_stub, &smooth_l1_kernel_cuda);
REGISTER_DISPATCH(sigmoid_backward_stub, &sigmoid_backward_kernel_cuda);
REGISTER_DISPATCH(tanh_backward_stub, &tanh_backward_kernel_cuda);
REGISTER_DISPATCH(mse_stub, &mse_kernel_cuda);

}} // namespace at::native
